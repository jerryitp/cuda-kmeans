#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_DEPRECATE

#include "math.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "k_mean.h"
#include <ctime>
#include <chrono>
#include <iostream>

#include "k_meanCUDA.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""

#define tile_size 32
#define numofcentr  4
#define numofelements 100000
#define numberofiter  800


void readfile(unit* read) {


	FILE* fid;



	if ((fid = fopen("dataset.txt", "r")) == NULL)
	{
		
		printf("Cannot open file");
		return;

	}
	
	char first[] = "firstMiddle";
	char sec[] = "secondMiddle";
	char thir[] = "thirdMiddle";
	char forth[] = "forthMiddle";

	char temp[20];

	for (int i = 0; i < numofelements;i++)
	{
		fscanf(fid, "%lf %lf %lf %lf %s\n", &read[i].dim1, &read[i].dim2, &read[i].dim3, &read[i].dim4, temp);
		//printf("%d\n", i);
		read[i].cluster = -1;
		if (strcmp(temp, first) == 0) {
			read[i].initcluster = 0;
		}
		if (strcmp(temp, sec) == 0) {
			read[i].initcluster = 1;
		}
		if (strcmp(temp, thir) == 0) {
			read[i].initcluster = 2;
		}
		if (strcmp(temp, forth) == 0) {
			read[i].initcluster = 3;
		}


	}

	fclose(fid);


}
void printdata(unit* data, unit* centroids ) {
	//for (int i = 0; i < numofelements; i++) {
	//	printf("%f %f %f %f ", data[i].dim1, data[i].dim2, data[i].dim3, data[i].dim4);
	//	printf("%d %d\n", data[i].cluster, data[i].initcluster);

	//}
	for (int i = 0; i < numofcentr;i++) {
		printf("\nCentroid #%d : %f %f %f %f ", i, centroids[i].dim1, centroids[i].dim2, centroids[i].dim3, centroids[i].dim4);
	}
	
}

void CPU_k_mean() {



	unit* data = (unit*)malloc(numofelements * sizeof(unit));
	unit* centroids = (unit*)calloc(numofcentr, sizeof(unit));


	readfile(data);

	centroids[0].dim1 = 0.5848632019399136;
	centroids[0].dim2 = 0.611966857319303;
	centroids[0].dim3 = 0.5976614560049508;
	centroids[0].dim4 = 0.6043396335008441;

	centroids[1].dim1 = 0.4057890614036615;
	centroids[1].dim2 = 0.39943930016009804 ;
	centroids[1].dim3 = 0.36002085029896843 ;
	centroids[1].dim4 = 0.3767179191381278;

	centroids[2].dim1 = 0.8436451075785734 ;
	centroids[2].dim2 = 0.8411071987518389 ;
	centroids[2].dim3 = 0.8192596107916138 ;
	centroids[2].dim4 = 0.8403777939339493;

	centroids[3].dim1 = 0.1571485475682983 ;
	centroids[3].dim2 = 0.16268196743699023 ;
	centroids[3].dim3 = 0.16264258613702254 ;
	centroids[3].dim4 = 0.16980746682214473;

	int i = 0;
	////
	//	Code for measuring was taken from http://www.informit.com/articles/article.aspx?p=1881386&seqNum=2
	//  Beginning
	//
	////
	std::chrono::time_point<std::chrono::system_clock> start, end;
	std::chrono::duration<double> elapsed_seconds;
	std::time_t end_time;

	start = std::chrono::system_clock::now();
	for (; i < numberofiter;i++) {

		for (int k = 0; k < numofelements; k++) {

			closestcentroid(&data[k], centroids, numofcentr);

			
		}
		calculateMean(data, centroids, numofcentr, numofelements);

	}
	//
		end = std::chrono::system_clock::now();
		elapsed_seconds = end - start;
		end_time = std::chrono::system_clock::to_time_t(end);
		std::cout << " finished computation at " << std::ctime(&end_time)
			<< "elapsed time: " << elapsed_seconds.count() << "s\n";
		////
		//
		//End of time measuring
		////


	printdata(data, centroids);
	printf("\n %d", i);
	
	//printdata(data);
	//calculateMean(data, centroids, numofcentr, numofelements);
	free(data);
	free(centroids);
}

void GPU_k_mean() {


	unit* data = (unit*)malloc(numofelements * sizeof(unit));

	unit* centroids = (unit*)calloc(numofcentr, sizeof(unit));

	unit* d_data;
	unit* d_centroids;
	hipMalloc((void**)&d_data, numofelements * sizeof(unit));
	hipMalloc((void**)&d_centroids, numofcentr* sizeof(unit));

	readfile(data);
	///initializeCentroids(centroids, numofcentr);
	centroids[0].dim1 = 0.5848632019399136;
	centroids[0].dim2 = 0.611966857319303;
	centroids[0].dim3 = 0.5976614560049508;
	centroids[0].dim4 = 0.6043396335008441;

	centroids[1].dim1 = 0.4057890614036615;
	centroids[1].dim2 = 0.39943930016009804;
	centroids[1].dim3 = 0.36002085029896843;
	centroids[1].dim4 = 0.3767179191381278;

	centroids[2].dim1 = 0.8436451075785734;
	centroids[2].dim2 = 0.8411071987518389;
	centroids[2].dim3 = 0.8192596107916138;
	centroids[2].dim4 = 0.8403777939339493;

	centroids[3].dim1 = 0.1571485475682983;
	centroids[3].dim2 = 0.16268196743699023;
	centroids[3].dim3 = 0.16264258613702254;
	centroids[3].dim4 = 0.16980746682214473;

	int i = 0;
	////
	//	Code for measuring was taken from http://www.informit.com/articles/article.aspx?p=1881386&seqNum=2
	//  Beginning
	//
	////
	std::chrono::time_point<std::chrono::system_clock> start, end;
	std::chrono::duration<double> elapsed_seconds;
	std::time_t end_time;

	start = std::chrono::system_clock::now();


	dim3 dimBlock(tile_size, tile_size, 1);
	dim3 dimGrid((sqrt(numofelements) + tile_size) / tile_size, (sqrt(numofelements) + tile_size) / tile_size,1);

	for (; i < numberofiter; i++) {

		hipMemcpy(d_data, data, numofelements * sizeof(unit), hipMemcpyHostToDevice);
		hipMemcpy(d_centroids, centroids, numofcentr * sizeof(unit), hipMemcpyHostToDevice);

		closestcentroidGPU <<< dimGrid, dimBlock >>>(d_data, d_centroids, numofcentr, numofelements);
		hipDeviceSynchronize();

		hipMemcpy(data, d_data, numofelements * sizeof(unit), hipMemcpyDeviceToHost);
		

		calculateMean(data, centroids, numofcentr, numofelements);
		hipFree(d_centroids);

	}
	
	end = std::chrono::system_clock::now();
	elapsed_seconds = end - start;
	end_time = std::chrono::system_clock::to_time_t(end);
	std::cout << " finished computation at " << std::ctime(&end_time)
		<< "elapsed time: " << elapsed_seconds.count() << "s\n";
	////
	//
	//End of time measuring
	////

	printdata(data, centroids);
	printf("\n %d", i);

	free(data);
	free(centroids);
	hipFree(d_data);
	hipFree(d_centroids);


}

void GPU_k_meanShared() {

	


	unit* sh_data = (unit*)malloc(numofelements * sizeof(unit));

	unit* sh_centroids = (unit*)calloc(numofcentr, sizeof(unit));

	unit* sh_d_data;
	unit* sh_d_centroids;
	hipMalloc((void**)&sh_d_data, numofelements * sizeof(unit));
	hipMalloc((void**)&sh_d_centroids, numofcentr* sizeof(unit));

	readfile(sh_data);
	///initializeCentroids(centroids, numofcentr);
	sh_centroids[0].dim1 = 0.5848632019399136;
	sh_centroids[0].dim2 = 0.611966857319303;
	sh_centroids[0].dim3 = 0.5976614560049508;
	sh_centroids[0].dim4 = 0.6043396335008441;

	sh_centroids[1].dim1 = 0.4057890614036615;
	sh_centroids[1].dim2 = 0.39943930016009804;
	sh_centroids[1].dim3 = 0.36002085029896843;
	sh_centroids[1].dim4 = 0.3767179191381278;

	sh_centroids[2].dim1 = 0.8436451075785734;
	sh_centroids[2].dim2 = 0.8411071987518389;
	sh_centroids[2].dim3 = 0.8192596107916138;
	sh_centroids[2].dim4 = 0.8403777939339493;

	sh_centroids[3].dim1 = 0.1571485475682983;
	sh_centroids[3].dim2 = 0.16268196743699023;
	sh_centroids[3].dim3 = 0.16264258613702254;
	sh_centroids[3].dim4 = 0.16980746682214473;

	int i = 0;
	////
	//	Code for measuring was taken from http://www.informit.com/articles/article.aspx?p=1881386&seqNum=2
	//  Beginning
	//
	////
	std::chrono::time_point<std::chrono::system_clock> start, end;
	std::chrono::duration<double> elapsed_seconds;
	std::time_t end_time;

	start = std::chrono::system_clock::now();


	dim3 dimBlock1(tile_size, tile_size, 1);
	dim3 dimGrid1((sqrt(numofelements) + tile_size) / tile_size, (sqrt(numofelements) + tile_size) / tile_size, 1);

	for (; i < numberofiter; i++) {

		hipMemcpy(sh_d_data, sh_data, numofelements * sizeof(unit), hipMemcpyHostToDevice);
		hipMemcpy(sh_d_centroids, sh_centroids, numofcentr * sizeof(unit), hipMemcpyHostToDevice);

		closestcentroidSharedGPU << < dimGrid1, dimBlock1 >> >(sh_d_data, sh_d_centroids, numofcentr, numofelements);
		hipDeviceSynchronize();

		hipMemcpy(sh_data, sh_d_data, numofelements * sizeof(unit), hipMemcpyDeviceToHost);


		calculateMean(sh_data, sh_centroids, numofcentr, numofelements);
		hipFree(sh_d_centroids);

	}
	//
	end = std::chrono::system_clock::now();
	elapsed_seconds = end - start;
	end_time = std::chrono::system_clock::to_time_t(end);
	std::cout << " finished computation at " << std::ctime(&end_time)
		<< "elapsed time: " << elapsed_seconds.count() << "s\n";
	////
	//
	//End of time measuring
	////


	printdata(sh_data, sh_centroids);
	printf("\n %d", i);

	free(sh_data);
	free(sh_centroids);
	hipFree(sh_d_data);
	hipFree(sh_d_centroids);
}

int main() {
	CPU_k_mean();
	//system("pause");
	printf("\n\n\n");
	//GPU_k_mean();
	printf("\n\n\n");
	//system("pause");
	GPU_k_meanShared();
	return 0;
}
